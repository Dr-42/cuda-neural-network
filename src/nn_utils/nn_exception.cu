#include "nn_utils/nn_exception.h"

void NNException::throwIfDeviceErrorsOccurred(const char* exception_message) {
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		std::cerr << error << ": " << exception_message;
		throw NNException(exception_message);
	}
}
